
#include <hip/hip_runtime.h>
// Code by xTyzen IV
extern "C" {
    __global__ void bruteforce_kernel(char *token1, int len, int* results) {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        long long total_combinations = 1;

        for (int i = 0; i < len; ++i) {
            total_combinations *= 256;
        }

        for (long long i = idx; i < total_combinations; i += blockDim.x * gridDim.x) {                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                  // Code by xTyzen IV
            long long remaining = i;                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                    // Code by xTyzen IV                                    
            for (int j = 0; j < len; ++j) {
                token1[j] = remaining % 256;
                remaining /= 256;
            }
            
            results[i] = 1;
        }
    }
}

// Code by xTyzen IV
